#include <hip/hip_runtime.h>
#include <stdio.h>
#include "kernels/basic_kernel.cu"

int main() {
    const int N = 1 << 20; // 1M elements
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy inputs to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel and measure time
    float milliseconds = KernelManager::launchKernel(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    bool success = true;
    for (int i = 0; i < N; i++) {
        if (h_C[i] != 3.0f) {
            success = false;
            break;
        }
    }

    printf("Kernel execution time: %f ms\n", milliseconds);
    printf("Test %s\n", success ? "PASSED" : "FAILED");

    // Cleanup
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
