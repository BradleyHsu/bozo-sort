#include "hip/hip_runtime.h"
#include "bogo_sort_matv2.cuh"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>
#include <mma.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <stdint.h>
#include <cute/tensor.hpp>
#include <cute/layout.hpp>

// #define DEBUG_PERMUTE
#define DEBUG_PRINT 
// #define DEBUG_SORT
// #define DEBUG_RANDOM
#define PERMUTE_MATRIX_HEIGHT 64
#define PERMUTATION_LENGTH 32
#define PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024 1024
#define PERMUTATION_VECTORS_64x32_FLAT_LENGTH_2048 2048
#define LOWER_ROW 512
#define NEXT_BLOCK 16

#define OUTER_WIDTH 16
#define INNER_DIM 16
#define OUTER_HEIGHT 16

// #define TOTAL_PERMUTATIONS 1000000 
#define TOTAL_PERMUTATIONS 10000000000
// #define TOTAL_PERMUTATIONS 10
#define CHECK_DONE_PERMUTATIONS 1000000

using namespace nvcuda;
using namespace std;
using namespace cute;

__global__ void bogo_sort_matv2(int* data, int size, int* output, int* block_permutation_counts) {
    __align__(256) extern __shared__ uint8_t permutation_matrix[PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024]; // two 32x32 arrays
    __align__(256) extern __shared__ uint8_t formatted_permutation_matrix[PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024];
    __align__(256) extern __shared__ uint8_t permutation_vectors[PERMUTATION_VECTORS_64x32_FLAT_LENGTH_2048];
    extern __shared__ int temp_permutation[PERMUTATION_LENGTH];

    extern __device__ int done;
    extern __shared__ int local_done;
    extern __shared__ bool is_sorted;

    if (threadIdx.x == 0) {
        local_done = 0;
        is_sorted = false;
        if (blockIdx.x == 0) {
            done = 0;
        }
    }
    __syncthreads();
    
    
    // Initialize random states and generate random ints
    extern __shared__ hiprandStatePhilox4_32_10_t random_states[PERMUTATION_LENGTH];
    extern __shared__ int random_ints[PERMUTATION_LENGTH];
    extern __shared__ long permutations_tried;

    
        
        //hiprand_init(unsigned long long seed,
        //  unsigned long long subsequence,
        //  unsigned long long offset,
        //  hiprandStatePhilox4_32_10_t *state)

    if (threadIdx.x < 32) {
        hiprand_init(blockIdx.x, threadIdx.x, 0, &random_states[threadIdx.x]);
        random_ints[threadIdx.x] = hiprand(&random_states[threadIdx.x]);
    }
    bogo_sort_permutation_gen(temp_permutation, size, random_ints);

    for (int i = 0; i < PERMUTE_MATRIX_HEIGHT; i++) {
        if (threadIdx.x < 32) {
            random_ints[threadIdx.x] = hiprand(&random_states[threadIdx.x]);
        }
        __syncthreads();
        bogo_sort_permutation_gen(temp_permutation, size, random_ints);
        if (threadIdx.x < 32) {
            permutation_vectors[i * PERMUTATION_LENGTH + threadIdx.x] = __float2half(data[temp_permutation[threadIdx.x]]);
        }
        __syncthreads();
    }
    
    if (threadIdx.x < 32) {
        random_ints[threadIdx.x] = hiprand(&random_states[threadIdx.x]);
    }
    __syncthreads();
    bogo_sort_basis_gen(permutation_matrix, size, random_ints);
    __syncthreads();
    if (threadIdx.x < 32) {
        random_ints[threadIdx.x] = hiprand(&random_states[threadIdx.x]);
    }
    __syncthreads();
    bogo_sort_basis_gen(permutation_matrix + PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024, size, random_ints);
    __syncthreads();

    #ifdef DEBUG_PRINT
    if (threadIdx.x == 0) {
        printf("Before Matmul Permutation vectors:\n");
        for (int i = 0; i < PERMUTE_MATRIX_HEIGHT; i++) {
            printf("  Row %2d: ", i);
            for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                if (i * PERMUTATION_LENGTH + j == 1772) {
                    print("[[1772]]");
                }
                printf("%3d ", permutation_vectors[i * PERMUTATION_LENGTH + j]);
            }
            printf("\n");
        }
        printf("\n");
    }
    __syncthreads();
    #endif

    extern __shared__ uint32_t switch_indexer;
    extern __shared__ uint32_t switch_multiplier;
    extern __shared__ uint32_t switch_incrementer;
    if (threadIdx.x == 0) {
        permutations_tried = 0;
        switch_indexer = hiprand(&random_states[threadIdx.x]);
        switch_incrementer = hiprand(&random_states[threadIdx.x]);
        switch_multiplier = switch_indexer;
    }
    __syncthreads();

    Tensor trivial_tensor = make_tensor(make_smem_ptr(permutation_matrix), make_shape(Int<32>{}, Int<32>{}));
    Tensor formatted_tensor = make_tensor(make_smem_ptr(formatted_permutation_matrix), make_shape(make_shape(Int<8>{}, Int<2>{}), make_shape(Int<16>{}, Int<4>{})),
        make_stride(make_stride(Int<16>{}, Int<1024>{}), make_stride(Int<1>{}, Int<8>{})));

    // cute::copy(trivial_tensor, formatted_tensor);
    cute::copy(trivial_tensor, formatted_tensor);

    // #ifdef DEBUG_PRINT
    // if (threadIdx.x == 0) {
    //     printf("Permutation Matrix:\n");
    //     for (int i = 0; i < 32; i++) {
    //         printf("  Row %2d: ", i);
    //         for (int j = 0; j < 32; j++) {
    //             printf("%u ", permutation_matrix[i * 32 + j]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");

    //     printf("Formatted Permutation Matrix:\n");
    //     for (int i = 0; i < 8; i++) {
    //         for (int j = 0; j < 2; j++) {
    //             printf("  Block (%d, %d):\n", i, j);
    //             for (int k = 0; k < 16; k++) {
    //                 printf("    Row %2d: ", k);
    //                 for (int l = 0; l < 4; l++) {
    //                     printf("%u ", formatted_permutation_matrix[i * 2 * 16 * 4 + j * 16 * 4 + k * 4 + l]);
    //                 }
    //                 printf("\n");
    //             }
    //         }
    //     }
    //     printf("\n");
    // }
    // __syncthreads();
    // #endif

    // if (threadIdx.x == 0) {
    //     printf("Permutation Matrix (linear):\n");
    //     for (int i = 0; i < 32 * 32; i++) {
    //         printf("%u ", permutation_matrix[i]);
    //         if ((i + 1) % 32 == 0) {
    //             printf("\n");
    //         }
    //     }
    //     printf("\n");

    //     printf("Formatted Permutation Matrix (linear):\n");
    //     for (int i = 0; i < 8 * 2 * 16 * 4; i++) {
    //         printf("%u ", formatted_permutation_matrix[i]);
    //         if ((i + 1) % (16 * 4) == 0) {
    //             printf("\n");
    //         }
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

    // #define ROW_LENGTH 8
    // #define CORE_MATRIX_HEIGHT 16
    // #define SHIFT_RIGHT 4
    // #define SHIFT_DOWN 64
    // #define WARP_SIZE 32
    //We have a 64 x 32 array of uint8s (num rows x bnum columns)
    //We split into 4 16 x 32 warps  
    //Each warp has 4  8 x 16 core matrices
    //
    //                                    32 columns
    //  +------------------------------------------------+
    //  |+----------------+----------------+              |
    //  ||    16x8       |     16x8       |              |
    //  |+----------------+----------------+     Warp 0    | 16 rows
    //  |+----------------+----------------+              |
    //  ||    16x8       |     16x8       |              |
    //  |+----------------+----------------+              |
    //  |------------------------------------------------|
    //  |+----------------+----------------+              |
    //  ||    16x8       |     16x8       |              |
    //  |+----------------+----------------+     Warp 1    | 16 rows
    //  |+----------------+----------------+              | 64 rows
    //  ||    16x8       |     16x8       |              |
    //  |+----------------+----------------+              |
    //  |------------------------------------------------|
    //  |+----------------+----------------+              |
    //  ||    16x8       |     16x8       |              |
    //  |+----------------+----------------+     Warp 2    | 16 rows
    //  |+----------------+----------------+              |
    //  ||    16x8       |     16x8       |              |
    //  |+----------------+----------------+              |
    //  |------------------------------------------------|
    //  |+----------------+----------------+              |
    //  ||    16x8       |     16x8       |              |
    //  |+----------------+----------------+     Warp 3    | 16 rows
    //  |+----------------+----------------+              |
    //  ||    16x8       |     16x8       |              |
    //  |+----------------+----------------+              |
    //  +------------------------------------------------+

    //each thread gets a vector of 4 .b32 registers
    //each register is packed with 4 u8 ints
    //each thread loads 4 ints from each of the 4 core matrices, to load 16 ints in all
    //each register gets 4 ints from the same core matrix

    // in terms of 8-bit words
    #define ROW_LENGTH 32
    #define CORE_MATRIX_HEIGHT 8
    #define CORE_MATRIX_ROW_LENGTH 16
    #define CORE_MATRICES_VERTICALLY_PER_WARP 2
    #define SHIFT_RIGHT 16
    #define SHIFT_DOWN 256
    #define WARP_SIZE 32

    // // in terms of 32-bit words
    // #define ROW_LENGTH 8
    // #define CORE_MATRIX_HEIGHT 16
    // #define SHIFT_RIGHT 4
    // #define SHIFT_DOWN 128
    // #define WARP_SIZE 32

    int warp_id = threadIdx.x / WARP_SIZE;
    int warp_address = warp_id * ROW_LENGTH * CORE_MATRIX_HEIGHT * 2;//two core matrixes vertically per warp

    #ifdef DEBUG_PRINT
    if (threadIdx.x == 0) {
        for (int i = 0; i < 4; i++) {
            printf("\nwarp address if warp_id =%d: %d \n", i, i * ROW_LENGTH * CORE_MATRIX_HEIGHT * 2);
        }
    }
    #endif 

    int global_idx = warp_id * ROW_LENGTH + (threadIdx.x % ROW_LENGTH);
    for (int i = 0; i < CORE_MATRIX_HEIGHT * 2; i++) {
        int offset = i * ROW_LENGTH;
        int val = (offset + global_idx) % 256;
        permutation_vectors[offset + global_idx] = __float2half(val);
    }
    __syncthreads();

    // if (threadIdx.x == 0) {
    //     printf("In thread register access (thread 0):\n");
    //         for (int i = 0; i < PERMUTE_MATRIX_HEIGHT; i++) {
    //             printf("  Row %2d: ", i);
    //             for (int j = 0; j < PERMUTATION_LENGTH; j++) {
    //                 printf("%2x ", permutation_vectors[i * PERMUTATION_LENGTH + j]);
    //             }
    //             printf("\n");
    //         }
    //         printf("\n");
    //     printf("warp_id: %d\n", warp_id);
    //     printf("warp_address: %d\n", warp_address);
    // }
    // __syncthreads();

    if (threadIdx.x == 127) {
        printf("In thread register access (thread 127):\n");
            for (int i = 0; i < PERMUTE_MATRIX_HEIGHT; i++) {
                printf("  Row %2d: ", i);
                for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                    printf("%2x ", permutation_vectors[i * PERMUTATION_LENGTH + j]);
                }
                printf("\n");
            }
            printf("\n");
    printf("warp_id: %d\n", warp_id);
    printf("warp_address: %d\n", warp_address);
    // typecast permutation vectors in terms of 32 bit words to ensure loads can be done
    uint32_t* permutation_vectors_32_bit = (uint32_t *) permutation_vectors;
    //each thread loads one contiguous 32 bit segment containing 4 u8 ints into a register in array
    //the threads are distributed to addresses in the 8 x 16 u8 as follows (8 x 4 b32)
    // Thread distribution in 8x4 matrix (row major order):
    //
    //       C0   C1   C2   C3   C4   C5   C6   C7
    //      +----+----+----+----+----+----+----+----+
    //  R0  | T0 | T1 | T2 | T3 | T0 | T1 | T2 | T3 |
    //      +----+----+----+----+----+----+----+----+
    //  R1  | T4 | T5 | T6 | T7 | T4 | T5 | T6 | T7 |
    //      +----+----+----+----+----+----+----+----+
    //  R2  | T8 | T9 |T10 |T11 | T8 | T9 |T10 |T11 |
    //      +----+----+----+----+----+----+----+----+
    //  R3  |T12 |T13 |T14 |T15 |T12 |T13 |T14 |T15 |
    //      +----+----+----+----+----+----+----+----+
    //  R4  |T16 |T17 |T18 |T19 |T16 |T17 |T18 |T19 |
    //      +----+----+----+----+----+----+----+----+
    //  R5  |T20 |T21 |T22 |T23 |T20 |T21 |T22 |T23 |
    //      +----+----+----+----+----+----+----+----+
    //  R6  |T24 |T25 |T26 |T27 |T24 |T25 |T26 |T27 |
    //      +----+----+----+----+----+----+----+----+
    //  R7  |T28 |T29 |T30 |T31 |T28 |T29 |T30 |T31 |
    //      +----+----+----+----+----+----+----+----+
    // Each of these rows is 4 b32s, but we want to index in u8s so we count 16 u8s per row
    int thread_core_matrix_row = (threadIdx.x % 32) / 4;
    int thread_core_matrix_col = (threadIdx.x % 32) % 4;
    int core_matrix_thread_location = thread_core_matrix_row * ROW_LENGTH + thread_core_matrix_col * 4;
    printf("thread_core_matrix_row: %d\n", thread_core_matrix_row);
    printf("thread_core_matrix_col: %d\n", thread_core_matrix_col);
    printf("core_matrix_thread_location: %d\n", core_matrix_thread_location);
    printf("thread assigned address %d\n", warp_address + core_matrix_thread_location);
    printf("u8 at address %d: %d\n", warp_address + core_matrix_thread_location, permutation_vectors[warp_address + core_matrix_thread_location]);

    for (int i = 0; i < 4; ++i) {
        int val = permutation_vectors[warp_address + i + core_matrix_thread_location];
        printf("u8 at %d, 0x%x\n",warp_address + i + core_matrix_thread_location, val);
        // result += val << (i * 8);
    }
    uint32_t bitf_0 = populate_bitf(permutation_vectors, warp_address + core_matrix_thread_location);
    uint32_t bitf_1 = populate_bitf(permutation_vectors, warp_address + core_matrix_thread_location + SHIFT_RIGHT);
    uint32_t bitf_2 = populate_bitf(permutation_vectors, warp_address + core_matrix_thread_location + SHIFT_DOWN);
    uint32_t bitf_3 = populate_bitf(permutation_vectors, warp_address + core_matrix_thread_location + SHIFT_DOWN + SHIFT_RIGHT);

    printf("bitf_0: 0x%08x\n", bitf_0);
    // printf("perm 128L : 0x%08x\n", permutation_vectors_32_bit[128]);
    printf("bitf_1: 0x%08x\n", bitf_1);
    printf("bitf_2: 0x%08x\n", bitf_2);
    printf("bitf_3: 0x%08x\n", bitf_3);

    // printf("Pretty Print of Permutation Vectors (32-bit):\n");
    // for (int i = 0; i < PERMUTE_MATRIX_HEIGHT; i++) {
    //     for (int j = 0; j < PERMUTATION_LENGTH / (sizeof(uint32_t) / sizeof(uint8_t)); j++) {
    //         printf("0x%08x ", permutation_vectors_32_bit[i * (PERMUTATION_LENGTH / (sizeof(uint32_t) / sizeof(uint8_t))) + j]);
    //     }
    //     printf("\n");
    // }

    printf("permutation_matrix pointer (hex): 0x%lx\n", (long)permutation_matrix);
    // printf("b_desc (decimal): %ld\n", b_desc);
    // printf("b_desc (hex): 0x%lx\n", b_desc);

    // printf("bitf_0: 0x%08x\n", bitf_0);
    // printf("bitf_1: 0x%08x\n", bitf_1); 
    // printf("bitf_2: 0x%08x\n", bitf_2);
    // printf("bitf_3: 0x%08x\n", bitf_3);

    __align__(256) extern __shared__ uint8_t output_vector_alloc[128];
    long output_vector = __cvta_generic_to_shared(output_vector_alloc);
    printf("output_vector (hex): %p\n", (void*)output_vector);
    printf("output_vector_alloc (hex): %p\n", (void*)output_vector_alloc);
    output_vector_alloc[0] = 0;
    output_vector_alloc[1] = 0;
    output_vector_alloc[2] = 0;
    output_vector_alloc[3] = 0;

    printf("output_1_addr: %p\n", (void*) output_vector_alloc);

    uint64_t permutation_matrix_start_address = (uint64_t) permutation_matrix;
    uint64_t b_desc = 0;
    // b_desc += matrix_descriptor_encode(permutation_matrix_start_address) << 0;
    // b_desc += 

    printf("permutation_matrix_start_address: %p\n", (void*)permutation_matrix_start_address);

    asm volatile (
                ".reg     .v4.b32 vec;\n"
                ".reg     .b32 r1;\n"

                "mov.b32 vec.x, %1;\n"
                "mov.b32 vec.y, %2;\n"
                "mov.b32 vec.z, %3;\n"
                "mov.b32 vec.w, %4;\n"

                // "st.shared.v4.b32 [%0], vec;\n"
                :
                :"l"(output_vector_alloc), "r"(bitf_0), "r"(bitf_1), "r"(bitf_2), "r"(bitf_3)
                : "memory");
    printf("Output vector: ");
    printf("Decimal: ");
    for (int i = 0; i < 16; i++) {
        printf("%02u ", output_vector_alloc[i]);
    }
    printf("\nHex: ");
    for (int i = 0; i < 16; i++) {
        printf("%02x ", output_vector_alloc[i]); 
    }
    printf("\n");
    }

    //we want to use wmma's native 16x16x16 matmul accumulate operations,
    //so each of our fragments needs to be 16x16x16
    //
    //we break down each 32x32 matrices into 16x16 tiles
    /*
                    32x32 Matrix Memory Layout
    
    +-------------------+-------------------+
    | ⬡                 | ⬢        ↑        |
    |<- NEXT_BLOCK=16 ->|          |        |
    |                   |  LOWER_ROW=16*32  |
    |                   |          |        |
    |                   |          ↓        |
    +-------------------+-------------------+
    | ⬣                 | ⬤                |
    |                   |                   |
    |                   |                   |
    |                   |                   |
    +-------------------+--------------------+

    Memory addresses for each quadrant:
    
    ⬡ ←── permutation_matrix
    ⬢ ←── permutation_matrix + NEXT_BLOCK
    ⬣ ←── permutation_matrix + LOWER_ROW  
    ⬤ ←── permutation_matrix + LOWER_ROW + NEXT_BLOCK

    Each 16x16 quadrant maps to tensor core fragments:
    ⬡ → mat_nw_frag     ⬢ → mat_ne_frag
    ⬣ → mat_sw_frag     ⬤ → mat_se_frag
    */
    /*
    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_ne_frag;
    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_nw_frag;
    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_se_frag;
    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_sw_frag;

    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_ne_alt_frag;
    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_nw_alt_frag;
    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_se_alt_frag;
    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_sw_alt_frag;

    wmma::fragment<wmma::matrix_b, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::col_major> vec_up_frag;
    wmma::fragment<wmma::matrix_b, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::col_major> vec_down_frag;

    wmma::fragment<wmma::accumulator, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half> prod_up_frag;
    wmma::fragment<wmma::accumulator, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half> prod_down_frag;

    wmma::load_matrix_sync(vec_up_frag, permutation_vectors, PERMUTATION_LENGTH);
    wmma::load_matrix_sync(vec_down_frag, permutation_vectors + NEXT_BLOCK, PERMUTATION_LENGTH);

    wmma::load_matrix_sync(mat_nw_frag, permutation_matrix, PERMUTATION_LENGTH);
    wmma::load_matrix_sync(mat_ne_frag, permutation_matrix + NEXT_BLOCK, PERMUTATION_LENGTH);
    wmma::load_matrix_sync(mat_sw_frag, permutation_matrix + LOWER_ROW, PERMUTATION_LENGTH);
    wmma::load_matrix_sync(mat_se_frag, permutation_matrix + LOWER_ROW + NEXT_BLOCK, PERMUTATION_LENGTH);

    wmma::load_matrix_sync(mat_nw_alt_frag, permutation_matrix + PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024, PERMUTATION_LENGTH);
    wmma::load_matrix_sync(mat_ne_alt_frag, permutation_matrix + NEXT_BLOCK + PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024, PERMUTATION_LENGTH);
    wmma::load_matrix_sync(mat_sw_alt_frag, permutation_matrix + LOWER_ROW + PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024, PERMUTATION_LENGTH);
    wmma::load_matrix_sync(mat_se_alt_frag, permutation_matrix + LOWER_ROW + NEXT_BLOCK + PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024, PERMUTATION_LENGTH);
    __syncthreads();
    */

    

    // asm volatile ("");

    // while (permutations_tried < TOTAL_PERMUTATIONS) {
    //     #ifdef DEBUG_RANDOM
    //     if (threadIdx.x == 0) {
    //         printf("Reached permutation %ld\n", permutations_tried);
    //         printf("switch_indexer:     ");
    //         for (int i = 31; i >= 0; i--) {
    //             printf("%d", (switch_indexer >> i) & 1);
    //             if (i % 8 == 0) printf(" ");
    //         }
    //         printf("\nswitch_incrementer: ");
    //         for (int i = 31; i >= 0; i--) {
    //             printf("%d", (switch_incrementer >> i) & 1);
    //             if (i % 8 == 0) printf(" ");
    //         }
    //         printf("\nswitch_multiplier:  ");
    //         for (int i = 31; i >= 0; i--) {
    //             printf("%d", (switch_multiplier >> i) & 1);
    //             if (i % 8 == 0) printf(" ");
    //         }
    //         printf("\nrandom_bit:         %s\n\n", random_bit ? "true" : "false");
    //     }
    //     #endif

    //         //get 16th-ish bit of switch_indexer, should be random bit
    //     // if ((switch_indexer >> 16) & 1) {
    //     //     wmma::fill_fragment(prod_up_frag, 0.0f);
    //     //     wmma::mma_sync(prod_up_frag, mat_nw_frag, vec_up_frag, prod_up_frag);
    //     //     wmma::mma_sync(prod_up_frag, mat_ne_frag, vec_down_frag, prod_up_frag);
    //     //     wmma::store_matrix_sync(permutation_vectors, prod_up_frag, PERMUTATION_LENGTH, wmma::mem_col_major);

    //     //     wmma::fill_fragment(prod_down_frag, 0.0f);
    //     //     wmma::mma_sync(prod_down_frag, mat_sw_frag, vec_up_frag, prod_down_frag);
    //     //     wmma::mma_sync(prod_down_frag, mat_se_frag, vec_down_frag, prod_down_frag);
    //     //     wmma::store_matrix_sync(permutation_vectors + NEXT_BLOCK, prod_down_frag, PERMUTATION_LENGTH, wmma::mem_col_major);
    //     // } else {
    //     //     wmma::fill_fragment(prod_up_frag, 0.0f);
    //     //     wmma::mma_sync(prod_up_frag, mat_nw_alt_frag, vec_up_frag, prod_up_frag);
    //     //     wmma::mma_sync(prod_up_frag, mat_ne_alt_frag, vec_down_frag, prod_up_frag);
    //     //     wmma::store_matrix_sync(permutation_vectors, prod_up_frag, PERMUTATION_LENGTH, wmma::mem_col_major);

    //     //     wmma::fill_fragment(prod_down_frag, 0.0f);
    //     //     wmma::mma_sync(prod_down_frag, mat_sw_alt_frag, vec_up_frag, prod_down_frag);
    //     //     wmma::mma_sync(prod_down_frag, mat_se_alt_frag, vec_down_frag, prod_down_frag);
    //     //     wmma::store_matrix_sync(permutation_vectors + NEXT_BLOCK, prod_down_frag, PERMUTATION_LENGTH, wmma::mem_col_major);
    //     // }



    //     /*
    //     wmma::load_matrix_sync(vec_up_frag, permutation_vectors, PERMUTATION_LENGTH);
    //     wmma::load_matrix_sync(vec_down_frag, permutation_vectors + NEXT_BLOCK, PERMUTATION_LENGTH);

    //     if (threadIdx.x == 0) {
    //         // these operations are weird because they are intended to
    //         //      1) generate a mostly random new bit
    //         //      2) in as few clock cycles + hits to shared memory as possible
    //         // can definitely be improved
    //         //
    //         // shift all bits in incrementer left by 1
    //         switch_incrementer = (switch_incrementer << 1) | (switch_incrementer >> 31);
    //         // do weird fucky-wucky operation to generate a psuedorandom new switch_indexer
    //         switch_indexer = switch_indexer * switch_multiplier + switch_incrementer;
    //         permutations_tried++;
    //         #ifdef DEBUG_RANDOM
    //         printf("Reached switch update at permutation %d with indexer=0x%08x, incrementer=0x%08x, multiplier=0x%08x\n", 
    //                permutations_tried, switch_indexer, switch_incrementer, switch_multiplier);
    //         printf("indexer: ");
    //         for (int i = 31; i >= 0; i--) {
    //             printf("%d", (switch_indexer >> i) & 1);
    //             if (i % 8 == 0) printf(" ");
    //         }
    //         printf("\n");
    //         #endif
    //     }

    //     __syncthreads();

    //     for (int i = 0; i < 16; i++) {
    //         //address of i'th row of permutation_vectors
    //         //assuming row major layout, and zero based indexing
    //         verify_sort_matv2(permutation_vectors + i * 32, 32, &is_sorted);
    //         if (is_sorted) {
    //             output[threadIdx.x] = permutation_vectors[i * 32 + threadIdx.x];
    //             if (threadIdx.x == 0) {
    //                 printf("Block %d found sorted array after %ld iterations and %ld permutations\n", blockIdx.x, permutations_tried, permutations_tried * 16);
    //                 atomicCAS(&done, 0, 1);
    //                 block_permutation_counts[blockIdx.x] = permutations_tried;
    //             }
    //             return;
    //         }
    //     }

    //     if (permutations_tried % CHECK_DONE_PERMUTATIONS == 1) {
    //         if (threadIdx.x == 0) {
    //             local_done = atomicAnd(&done, 1);
    //         }
    //         __syncthreads();
    //         if (local_done) {
    //             if (blockIdx.x%1000 == 0 && threadIdx.x == 0) {
    //                 printf("Block %d: Loops: %ld; Permutations tried: %ld\n", blockIdx.x, permutations_tried, permutations_tried * 16);
    //             }
    //             if (threadIdx.x == 0) {
    //                 block_permutation_counts[blockIdx.x] = permutations_tried;
    //             }
    //             return;
    //         }
    //     }
    //     */
    // }

    // #ifdef DEBUG_PRINT
    // if (threadIdx.x == 0) {
    //     printf("After Matmul Permutation vectors:\n");
    //     for (int i = 0; i < PERMUTE_MATRIX_HEIGHT; i++) {
    //         printf("  Row %2d: ", i);
    //         for (int j = 0; j < PERMUTATION_LENGTH; j++) {
    //             printf("%3d ", permutation_vectors[i * PERMUTATION_LENGTH + j]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");

    //     printf("Output data: ");
    //     for (int i = 0; i < size; i++) {
    //         printf("%3d ", data[i]);
    //     }
    //     printf("\n");

    //     printf("Total permutations tried: %ld\n", permutations_tried);
    // }
    // #endif

    return;
}

__device__ static inline uint32_t populate_bitf(uint8_t* array, int address) {
    uint32_t result = 0;
    for (int i = 0; i < 4; ++i) {
        int val = array[i + address];
        printf("0x%x\n", val);
        result += val << (i * 8);
    }
    return result;
}


__device__ static inline uint64_t matrix_descriptor_encode(uint64_t x) { return (((x) & 0x3FFFF) >> 0x4); }



__device__ void verify_sort_matv2(uint8_t* input, int size, bool* is_sorted) {
    __syncthreads();
    if (threadIdx.x == 0) {
        *is_sorted = true;
    }
    __syncthreads();
    if (threadIdx.x < size - 1 && threadIdx.x < 32) {  // Don't check the last element since it has no right neighbor
        if (input[threadIdx.x] > input[threadIdx.x + 1]) {
            *is_sorted = false;
        }
    }
    __syncthreads();
}

__device__ void bogo_sort_basis_gen(uint8_t* data, int size, int* random_ints) {
    extern __shared__ int sorted_ints[PERMUTATION_LENGTH * 2];
    auto parity_shift = [](int p) { return p ? PERMUTATION_LENGTH : 0;};
    __syncthreads();
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Random ints: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", random_ints[i]);
        }
        printf("\n");
    }
    __syncthreads();
    #endif
    
    // Copy random ints to sorted_ints initial parity section
    if (threadIdx.x < 32) {
        sorted_ints[threadIdx.x] = random_ints[threadIdx.x];
    }
    __syncthreads();

    extern __shared__ int step_size;
    extern __shared__ bool parity;
    extern __shared__ int merge_indices[PERMUTATION_LENGTH];
    if (threadIdx.x == 0) {
        step_size = 2;
        parity = false;
    }
    __syncthreads();

    while (step_size <= PERMUTATION_LENGTH) {
        if (threadIdx.x % step_size == 0 && threadIdx.x < 32) {
            int left_merge_counter = threadIdx.x;
            int right_merge_counter = threadIdx.x + 1;
            #ifdef DEBUG_SORT
            printf("Thread %d: left_merge_counter=%d, right_merge_counter=%d\n", 
                   threadIdx.x, left_merge_counter, right_merge_counter);
            #endif

            merge_indices[left_merge_counter] = 0;
            merge_indices[right_merge_counter] = 0;
            int print_thread_idx = 0;
            for (int i=0; i < step_size; i++) {
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("Thread %d, Step %d, Iteration %d:\n", threadIdx.x, step_size, i);
                }
                #endif
                
                int left_idx = threadIdx.x + merge_indices[left_merge_counter] + parity_shift(parity);
                int right_idx = threadIdx.x + merge_indices[right_merge_counter] + step_size/2 + parity_shift(parity);
                int dest_idx = threadIdx.x + i + parity_shift(!parity);
                
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("  Left index: %d (value: %d)\n", left_idx, sorted_ints[left_idx]);
                    printf("  Right index: %d (value: %d)\n", right_idx, sorted_ints[right_idx]);
                    printf("  Destination index: %d\n", dest_idx);
                }
                #endif
                
                bool take_from_left = merge_indices[right_merge_counter] == step_size/2 ||
                    (sorted_ints[left_idx] < sorted_ints[right_idx] && 
                     merge_indices[left_merge_counter] < step_size/2);
                
                if (take_from_left) {
                    sorted_ints[dest_idx] = sorted_ints[left_idx];
                    merge_indices[left_merge_counter]++;
                    #ifdef DEBUG_SORT
                    if (threadIdx.x == print_thread_idx) {
                        printf("  Taking from left array\n");
                    }
                    #endif
                } else {
                    sorted_ints[dest_idx] = sorted_ints[right_idx]; 
                    merge_indices[right_merge_counter]++;
                    #ifdef DEBUG_SORT
                    if (threadIdx.x == print_thread_idx) {
                        printf("  Taking from right array\n");
                    }
                    #endif
                }
                
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("  New value at destination: %d\n", sorted_ints[dest_idx]);
                    printf("  Left merge index: %d, Right merge index: %d\n\n", 
                           merge_indices[left_merge_counter], merge_indices[right_merge_counter]);
                    printf("  Current array state: ");
                    for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                        printf("%d ", sorted_ints[j + parity_shift(!parity)]);
                    }
                    printf("\n");
                }
                #endif
            }
        }
        if (threadIdx.x == 0) {
            step_size *= 2;
            parity = !parity;
        }
        __syncthreads();
        #ifdef DEBUG_SORT
        if (threadIdx.x == 0) {
            printf("Random ints after step %d: ", step_size);
            for (int i = 0; i < PERMUTATION_LENGTH; i++) {
                printf("%d ", sorted_ints[i + parity_shift(parity)]);
            }
            printf("\n");
        }
        __syncthreads();
        #endif
        
    }
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Sorted random ints: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", sorted_ints[i + parity_shift(parity)]);
        }
        printf("\n");
    }
    #endif
    __syncthreads();
    int my_value = random_ints[threadIdx.x];
    int final_index = -1;
    if (threadIdx.x < 32) {
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            if (sorted_ints[i + parity_shift(parity)] == my_value) {
                final_index = i;
                break;
            }
        }
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            data[threadIdx.x * PERMUTATION_LENGTH + i] = (i == final_index) ? 1 : 0;
        }
    }
    __syncthreads();

    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Final indices matrix:\n");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("  ");
            for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                printf("%d ", data[i * PERMUTATION_LENGTH + j]);
            }
            printf("\n");
        }
        printf("\n");
    }
    #endif

    // data[threadIdx.x] = sorted_ints[threadIdx.x + parity_shift(!parity)];
    __syncthreads();
}

__device__ void bogo_sort_permutation_gen(int* data, int size, int* random_ints) {
    extern __shared__ int sorted_ints[64];
    auto parity_shift = [](int p) { return p ? PERMUTATION_LENGTH : 0;};
    
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Random ints: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", random_ints[i]);
        }
        printf("\n");
    }
    __syncthreads();
    #endif
    
    // Copy random ints to sorted_ints initial parity section
    sorted_ints[threadIdx.x] = random_ints[threadIdx.x];
    extern __shared__ int step_size;
    extern __shared__ bool parity;
    extern __shared__ int merge_indices[PERMUTATION_LENGTH];
    if (threadIdx.x == 0) {
        step_size = 2;
        parity = false;
    }
    __syncthreads();
    while (step_size <= PERMUTATION_LENGTH) {
        if (threadIdx.x % step_size == 0 && threadIdx.x < 32) {
            int left_merge_counter = threadIdx.x;
            int right_merge_counter = threadIdx.x + 1;
            #ifdef DEBUG_SORT
            printf("Thread %d: left_merge_counter=%d, right_merge_counter=%d\n", 
                   threadIdx.x, left_merge_counter, right_merge_counter);
            #endif
            merge_indices[left_merge_counter] = 0;
            merge_indices[right_merge_counter] = 0;
            int print_thread_idx = 0;
            for (int i=0; i < step_size; i++) {
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("Thread %d, Step %d, Iteration %d:\n", threadIdx.x, step_size, i);
                }
                #endif
                
                int left_idx = threadIdx.x + merge_indices[left_merge_counter] + parity_shift(parity);
                int right_idx = threadIdx.x + merge_indices[right_merge_counter] + step_size/2 + parity_shift(parity);
                int dest_idx = threadIdx.x + i + parity_shift(!parity);
                
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("  Left index: %d (value: %d)\n", left_idx, sorted_ints[left_idx]);
                    printf("  Right index: %d (value: %d)\n", right_idx, sorted_ints[right_idx]);
                    printf("  Destination index: %d\n", dest_idx);
                }
                #endif
                
                bool take_from_left = merge_indices[right_merge_counter] == step_size/2 ||
                    (sorted_ints[left_idx] < sorted_ints[right_idx] && 
                     merge_indices[left_merge_counter] < step_size/2);
                
                if (take_from_left) {
                    sorted_ints[dest_idx] = sorted_ints[left_idx];
                    merge_indices[left_merge_counter]++;
                    #ifdef DEBUG_SORT
                    if (threadIdx.x == print_thread_idx) {
                        printf("  Taking from left array\n");
                    }
                    #endif
                } else {
                    sorted_ints[dest_idx] = sorted_ints[right_idx]; 
                    merge_indices[right_merge_counter]++;
                    #ifdef DEBUG_SORT
                    if (threadIdx.x == print_thread_idx) {
                        printf("  Taking from right array\n");
                    }
                    #endif
                }
                
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("  New value at destination: %d\n", sorted_ints[dest_idx]);
                    printf("  Left merge index: %d, Right merge index: %d\n\n", 
                           merge_indices[left_merge_counter], merge_indices[right_merge_counter]);
                    printf("  Current array state: ");
                    for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                        printf("%d ", sorted_ints[j + parity_shift(!parity)]);
                    }
                    printf("\n");
                }
                #endif
            }
        }
        if (threadIdx.x == 0) {
            step_size *= 2;
            parity = !parity;
        }
        __syncthreads();
        #ifdef DEBUG_SORT
        if (threadIdx.x == 0) {
            printf("Random ints after step %d: ", step_size);
            for (int i = 0; i < PERMUTATION_LENGTH; i++) {
                printf("%d ", sorted_ints[i + parity_shift(parity)]);
            }
            printf("\n");
        }
        __syncthreads();
        #endif
    }
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Sorted random ints: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", sorted_ints[i + parity_shift(parity)]);
        }
        printf("\n");
    }
    __syncthreads();
    #endif
    int my_value = random_ints[threadIdx.x];
    int my_index = -1;
    if (threadIdx.x < 32) {
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            if (sorted_ints[i + parity_shift(parity)] == my_value) {
                my_index = i;
                break;
            }
        }
        data[threadIdx.x] = my_index;
    }
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Final sorted array: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", data[i]);
        }
        printf("\n");
    }
    #endif
    __syncthreads();
}


dim3 KernelManagerBogoSortMatV2::calculateGrid(int n, int threadsPerBlock) {
    // return dim3((INNER_DIM + threadsPerBlock - 1) / threadsPerBlock);
    return dim3(n);
}

float KernelManagerBogoSortMatV2::launchKernel(int* data, int* output) {
    int size = 32;
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    int smCount = 0;
    for (int i = 0; i < deviceCount; i = i + 1) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        smCount += prop.multiProcessorCount;
    }

    #ifdef DEBUG_PRINT
    printf("Number of SMs: %d\n", smCount);
    #endif

    int numBlocks = smCount * 64;
    printf("Number of blocks: %d\n", numBlocks);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int threadsPerBlock = 128;
    dim3 grid = calculateGrid(numBlocks, threadsPerBlock);
    dim3 block(threadsPerBlock);
    #ifdef DEBUG_PRINT
    printf("Grid dimensions: %d x %d x %d\n", grid.x, grid.y, grid.z);
    #endif

    // Record start time
    hipEventRecord(start);

    // Allocate device memory for block permutation counts
    int* block_permutation_counts;
    hipMalloc(&block_permutation_counts, grid.x * sizeof(int));

    // Launch kernel
    // bogo_sort_matv2<<<grid, block>>>(data, size, output, block_permutation_counts);
    bogo_sort_matv2<<<1, block>>>(data, size, output, block_permutation_counts);
    // bogo_sort_matv2<<<1, block>>>(data, size, output);
    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy block permutation counts back to host and print
    int* h_block_permutation_counts = new int[grid.x];
    hipMemcpy(h_block_permutation_counts, block_permutation_counts, grid.x * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nBlock permutation counts:\n");
    for (int i = 0; i < grid.x; i=i+1000) {
        if (h_block_permutation_counts[i] > 0) {
            printf("Block %d: %d permutations\n", i, h_block_permutation_counts[i]);
        }
    }

    // Calculate and print total permutations across all blocks
    long total_block_permutations = 0;
    for (int i = 0; i < grid.x; i++) {
        total_block_permutations += h_block_permutation_counts[i];
    }
    printf("\nTotal block cycles computed: %'ld\n", total_block_permutations);

    // Each block permutation generates 16 actual permutations
    long total_permutations = total_block_permutations * 16;
    printf("Total actual permutations tried: %'ld\n", total_permutations);

    // Calculate FLOPS:
    // For each permutation:
    // - 16x16 matrix multiplied by 16x1 vector requires:
    //   16 rows * 16 columns * 2 operations (multiply + add) = 512 FLOPs per permutation
    double total_tflops = (total_permutations * 512.0) / 1e12;
    printf("Total TFLOPs performed: %.2f teraflops\n", total_tflops);

    // Cleanup
    delete[] h_block_permutation_counts;
    hipFree(block_permutation_counts);


    return milliseconds;
}
