#include "hip/hip_runtime.h"
#include "bogo_sort.cuh"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>

__global__ void bogo_sort(int* data, int size, int* output) {
    extern __shared__ int shared_data[64];
    extern __shared__ hiprandStatePhilox4_32_10_t shared_random_state;
    extern __shared__ int shared_random;

    extern __device__ int done;
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        done = 0;
    }
    extern __shared__ int local_done;
    if (threadIdx.x == 0) {
        local_done = 0;
    }

    extern __shared__ bool parity;
    parity = false;
    __shared__ bool is_sorted;
    if (threadIdx.x == 0) {
        is_sorted = false;
    }

     //hiprand_init(unsigned long long seed,
            // unsigned long long subsequence,
            // unsigned long long offset,
            // hiprandStatePhilox4_32_10_t *state)    
    hiprand_init(0, blockIdx.x, 0, &shared_random_state);
    __syncthreads();

    auto parity_shift = [](int p) { return p ? 32 : 0;};

    extern __shared__ long permutation;

    if (threadIdx.x == 0) {
        permutation = 0;
    }

    if (threadIdx.x < size) {
        shared_data[threadIdx.x] = data[threadIdx.x];
    }
    __syncthreads();

    do {
        shared_random = hiprand(&shared_random_state);
        // int idx = blockIdx.x * blockDim.x + threadIdx.x;

        // if (threadIdx.x == 0) {
        //     printf("shared_data after loading: ");
        //     for (int i = 0; i < 64; i++) {
        //         printf("%d ", shared_data[i]);
        //     }
        //     printf("\n");
        // }

        // #define DEBUG_ITERATIONS 1
        // #if DEBUG_ITERATIONS
        // if (blockIdx.x == 0)  {
        //     if (threadIdx.x == 0) {
        //         permutation++;
        //     }
        //     if (threadIdx.x == 0 && permutation % 100000 == 0) {
        //         printf("Iteration %ld, shared_data: ", permutation);
        //         for (int i = 0; i < size; i++) {
        //             printf("%d ", shared_data[i + parity_shift(parity)]);
        //         }
        //         printf("\n");
        //     }
        // }
        // #endif

        // bit shift right by threadIdx.x/2 bits, then 
        // check for parity to check if the threadIdx.x/2'th bit is 0 or 1
        // if (threadIdx.x == 0) {
        //     printf("\nThread operations:\n");
        // }
        if (threadIdx.x == 0) {
            permutation++;
        }
        int __shared__ offset;
        if (threadIdx.x == 0) {
            offset = 0;
        }
        int __shared__ swap_length;
        for (swap_length = 2; swap_length <= 32; swap_length *= 2) {
            if ((shared_random >> (threadIdx.x / swap_length + offset)) % 2 == 1) {
                if (threadIdx.x % swap_length < swap_length / 2) {
                    shared_data[threadIdx.x + swap_length / 2 + parity_shift(!parity)] = shared_data[threadIdx.x + parity_shift(parity)];
                } else {
                    shared_data[threadIdx.x - swap_length / 2 + parity_shift(!parity)] = shared_data[threadIdx.x + parity_shift(parity)];
                }
            } else {
                shared_data[threadIdx.x + parity_shift(!parity)] = shared_data[threadIdx.x + parity_shift(parity)]; 
            }

            if (threadIdx.x == 0) {
                offset += 32 / swap_length;
                parity = !parity;
            }
            __syncthreads();
        }

        if (threadIdx.x != 0) {
            shared_data[threadIdx.x + parity_shift(!parity)] = shared_data[threadIdx.x + parity_shift(parity) - 1];
        } else {
            shared_data[parity_shift(!parity)] = shared_data[size + parity_shift(parity) - 1]; 
            parity = !parity;
        }
        

        // if (threadIdx.x == 0) {
        //     printf("shared_random binary: ");
        //     for (int i = 31; i >= 0; i--) {
        //         printf("%d:%d ", i, (shared_random >> i) & 1);
        //     }
        //     printf("\n");
        //     printf("shared_data: ");
        //     for (int i = 0; i < 64; i++) {
        //         printf("%d ", shared_data[i]);
        //     }
        //     printf("\n");
        // }
        // __syncthreads();

        __syncthreads();
        verify_sort(shared_data + parity_shift(parity), 32, &is_sorted);
        __syncthreads();

        if (threadIdx.x == 0) {
            local_done = atomicAnd(&done, 1);
        }
        __syncthreads();
        if (local_done != 0) {
            if (threadIdx.x == 0 && blockIdx.x % 500 == 0) {
                printf("Block %d has tried %ld to end\n", blockIdx.x, permutation);
            }
            return;
        }
        __syncthreads();
    } while (!is_sorted);

    __syncthreads();
    output[threadIdx.x] = shared_data[threadIdx.x + parity_shift(parity)];
    if (threadIdx.x == 0) {
        printf("Before setting: %d\n", done);
        printf("Block %d found sorted array after %ld permutations\n", blockIdx.x, permutation);
        atomicCAS(&done, 0, 1);
        printf("After setting: %d\n", done);
    }
    __syncthreads();
    return;
}

__device__ void verify_sort(int* input, int size, bool* is_sorted) {
    __syncthreads();
    if (threadIdx.x == 0) {
        *is_sorted = true;
    }
    __syncthreads();
    if (threadIdx.x < size - 1) {  // Don't check the last element since it has no right neighbor
        if (input[threadIdx.x] > input[threadIdx.x + 1]) {
            *is_sorted = false;
        }
    }
    __syncthreads();
}

// __global__ void bogo_sort_working(int* data, int size, int* output) {
//     extern __shared__ int shared_data[64];
//     extern __shared__ hiprandStatePhilox4_32_10_t shared_random_state;
//     extern __shared__ int shared_random;

//     extern __shared__ bool parity;
//     parity = false;
//     __syncthreads();

//     auto parity_shift = [](int p) { return p ? 32 : 0;};
    
//     //hiprand_init(unsigned long long seed,
//             // unsigned long long subsequence,
//             // unsigned long long offset,
//             // hiprandStatePhilox4_32_10_t *state)
//     hiprand_init(0, 0, 0, &shared_random_state);
//     shared_random = hiprand(&shared_random_state);
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx < size) {
//         output[idx] = shared_random;
//     }

//     if (threadIdx.x < 32 && threadIdx.x < size) {
//         shared_data[threadIdx.x] = data[threadIdx.x];
//     }
//     __syncthreads();
//     // if (threadIdx.x == 0) {
//     //     printf("shared_data after loading: ");
//     //     for (int i = 0; i < 64; i++) {
//     //         printf("%d ", shared_data[i]);
//     //     }
//     //     printf("\n");
//     // }


//     // bit shift right by threadIdx.x/2 bits, then 
//     // check for parity to check if the threadIdx.x/2'th bit is 0 or 1
//     // if (threadIdx.x == 0) {
//     //     printf("\nThread operations:\n");
//     // }

//     int __shared__ offset;
//     if (threadIdx.x == 0) {
//         offset = 0;
//     }
//     int __shared__ swap_length;
//     for (swap_length = 2; swap_length <= 32; swap_length *= 2) {
//         if ((shared_random >> (threadIdx.x / swap_length + offset)) % 2 == 1) {
//             if (threadIdx.x % swap_length < swap_length / 2) {
//                 shared_data[threadIdx.x + swap_length / 2 + parity_shift(!parity)] = shared_data[threadIdx.x + parity_shift(parity)];
//             } else {
//                 shared_data[threadIdx.x - swap_length / 2 + parity_shift(!parity)] = shared_data[threadIdx.x + parity_shift(parity)];
//             }
//         } else {
//             shared_data[threadIdx.x + parity_shift(!parity)] = shared_data[threadIdx.x + parity_shift(parity)]; 
//         }

//         if (threadIdx.x == 0) {
//             offset += 32 / swap_length;
//             parity = !parity;
//         }
//         __syncthreads();
//     }

//     if (threadIdx.x != 0) {
//         shared_data[threadIdx.x + parity_shift(!parity)] = shared_data[threadIdx.x + parity_shift(parity) - 1];
//     } else {
//         shared_data[parity_shift(!parity)] = shared_data[size + parity_shift(parity) - 1]; 
//         parity = !parity;
//     }
//     __syncthreads();

//     // if (threadIdx.x == 0) {
//     //     printf("shared_random binary: ");
//     //     for (int i = 31; i >= 0; i--) {
//     //         printf("%d:%d ", i, (shared_random >> i) & 1);
//     //     }
//     //     printf("\n");
//     //     printf("shared_data: ");
//     //     for (int i = 0; i < 64; i++) {
//     //         printf("%d ", shared_data[i]);
//     //     }
//     //     printf("\n");
//     // }
//     // __syncthreads();

//     output[threadIdx.x] = shared_data[threadIdx.x + parity_shift(parity)];        
//     __shared__ bool is_sorted;
//     if (threadIdx.x == 0) {
//         is_sorted = true;
//     }
//     __syncthreads();

//     verify_sort(shared_data + parity_shift(parity), 32, &is_sorted);
//     __syncthreads();
        
// }

dim3 KernelManagerBogoSort::calculateGrid(int N, int threadsPerBlock) {
    // return dim3((N + threadsPerBlock - 1) / threadsPerBlock);
    return dim3(N);
}

float KernelManagerBogoSort::launchKernel(int* data, int* output) {
    int size = 32;
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    int smCount = 0;
    for (int i = 0; i < deviceCount; i = i + 1) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        smCount += prop.multiProcessorCount;
    }

    printf("Number of SMs: %d\n", smCount);

    int numBlocks = smCount * 32;
    printf("Number of blocks: %d\n", numBlocks);


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int threadsPerBlock = 32;
    dim3 grid = calculateGrid(numBlocks, threadsPerBlock);
    dim3 block(threadsPerBlock);
    printf("Grid dimensions: %d x %d x %d\n", grid.x, grid.y, grid.z);

    // Record start time
    hipEventRecord(start);

    // Launch kernel
    bogo_sort<<<grid, block>>>(data, size, output);
    // bogo_sort<<<100, block>>>(data, size, output);

    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds;
}
