#include "hip/hip_runtime.h"
#include "bogo_sort_matv1.cuh"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>
#include <mma.h>
#include <hip/hip_runtime.h>

#define DEBUG_PERMUTE
#define DEBUG_PRINT 
// #define DEBUG_SORT 
#define PERMUTE_MATRIX_WIDTH 16
#define PERMUTATION_LENGTH 32
#define PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024 1024
#define PERMUTATION_ARRAY_32x16_FLAT_LENGTH_512 512

// \begin{courtesy of Zong-Sheng Wang}
#define M 16
#define N 16
#define K 16
// \end{courtesy of Zong-Sheng Wang}

__global__ void bogo_sort_matv1(int* data, int size, int* output) {
    extern __shared__ int permutation_matrix[PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024]; // 32x32 array
    extern __shared__ int permutation_array[PERMUTATION_ARRAY_32x16_FLAT_LENGTH_512];
    extern __shared__ int temp_permutation[PERMUTATION_LENGTH];

    #ifdef DEBUG_PERMUTE
    for (int i = 0; i < PERMUTE_MATRIX_WIDTH; i++) {
        permutation_array[i * PERMUTATION_LENGTH + threadIdx.x] = threadIdx.x;
    }
    __syncthreads();
    #endif
    
    // Initialize random states and generate random ints
    extern __shared__ hiprandStatePhilox4_32_10_t random_states[PERMUTATION_LENGTH];
    extern __shared__ int random_ints[PERMUTATION_LENGTH];
    hiprand_init(0, threadIdx.x, 0, &random_states[threadIdx.x]);
    __syncthreads();
    
    for (int i = 0; i < PERMUTE_MATRIX_WIDTH; i++) {
        random_ints[threadIdx.x] = hiprand(&random_states[threadIdx.x]);
        __syncthreads();
        bogo_sort_permutation_gen(temp_permutation, size, random_ints);
        __syncthreads();
        
        // Each thread copies its value to the right spot in the 512 array
        if (threadIdx.x < PERMUTATION_LENGTH) {
            permutation_array[i * PERMUTATION_LENGTH + threadIdx.x] = data[temp_permutation[threadIdx.x]];
        }
        __syncthreads();
    }

    random_ints[threadIdx.x] = hiprand(&random_states[threadIdx.x]);
    __syncthreads();
    bogo_sort_basis_gen(permutation_matrix, size, random_ints);

    // \begin{courtesy of Zong-Sheng Wang}
    wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major> a_frag;
	wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major> b_frag;
	wmma::fragment<wmma::accumulator, M, N, K, float> ab_frag;

    wmma::fill_fragment(ab_frag, 0.0f);

    wmma::load_matrix_sync(a_frag, permutation_matrix, K);
    wmma::load_matrix_sync(b_frag, permutation_matrix, K);
    wmma::mma_sync(ab_frag, a_frag, b_frag, ab_frag);
    // \end{courtesy of Zong-Sheng Wang}

    #ifdef DEBUG_PRINT
    if (threadIdx.x == 0) {
        printf("Permutation array:\n");
        for (int i = 0; i < PERMUTE_MATRIX_WIDTH; i++) {
            printf("  Row %2d: ", i);
            for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                printf("%2d ", permutation_array[i * PERMUTATION_LENGTH + j]);
            }
            printf("\n");
        }
        printf("\n");

        printf("Permutation matrix:\n");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("  Row %2d: ", i);
            for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                printf("%d ", permutation_matrix[i * PERMUTATION_LENGTH + j]);
            }
            printf("\n");
        }
        printf("\n");
    }
    #endif
    __syncthreads();
    #ifdef DEBUG_PRINT
    if (threadIdx.x == 0) {
        printf("Output data: ");
        for (int i = 0; i < size; i++) {
            printf("%d ", data[i]);
        }
        printf("\n");
    }
    #endif
    return;
}

__device__ void verify_sort_matv1(int* input, int size, bool* is_sorted) {
    __syncthreads();
    if (threadIdx.x == 0) {
        *is_sorted = true;
    }
    __syncthreads();
    if (threadIdx.x < size - 1) {  // Don't check the last element since it has no right neighbor
        if (input[threadIdx.x] > input[threadIdx.x + 1]) {
            *is_sorted = false;
        }
    }
    __syncthreads();
}

__device__ void bogo_sort_basis_gen(int* data, int size, int* random_ints) {
    extern __shared__ int sorted_ints[PERMUTATION_LENGTH * 2];
    auto parity_shift = [](int p) { return p ? PERMUTATION_LENGTH : 0;};
    
    __syncthreads();
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Random ints: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", random_ints[i]);
        }
        printf("\n");
    }
    #endif
    __syncthreads();

    // Copy random ints to sorted_ints initial parity section
    sorted_ints[threadIdx.x] = random_ints[threadIdx.x];
    __syncthreads();

    extern __shared__ int step_size;
    extern __shared__ bool parity;
    extern __shared__ int merge_indices[PERMUTATION_LENGTH];
    if (threadIdx.x == 0) {
        step_size = 2;
        parity = false;
    }
    __syncthreads();

    while (step_size <= PERMUTATION_LENGTH) {
        if (threadIdx.x % step_size == 0) {
            int left_merge_counter = threadIdx.x;
            int right_merge_counter = threadIdx.x + 1;
            #ifdef DEBUG_SORT
            printf("Thread %d: left_merge_counter=%d, right_merge_counter=%d\n", 
                   threadIdx.x, left_merge_counter, right_merge_counter);
            #endif

            merge_indices[left_merge_counter] = 0;
            merge_indices[right_merge_counter] = 0;
            int print_thread_idx = 0;
            for (int i=0; i < step_size; i++) {
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("Thread %d, Step %d, Iteration %d:\n", threadIdx.x, step_size, i);
                }
                #endif
                
                int left_idx = threadIdx.x + merge_indices[left_merge_counter] + parity_shift(parity);
                int right_idx = threadIdx.x + merge_indices[right_merge_counter] + step_size/2 + parity_shift(parity);
                int dest_idx = threadIdx.x + i + parity_shift(!parity);
                
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("  Left index: %d (value: %d)\n", left_idx, sorted_ints[left_idx]);
                    printf("  Right index: %d (value: %d)\n", right_idx, sorted_ints[right_idx]);
                    printf("  Destination index: %d\n", dest_idx);
                }
                #endif
                
                bool take_from_left = merge_indices[right_merge_counter] == step_size/2 ||
                    (sorted_ints[left_idx] < sorted_ints[right_idx] && 
                     merge_indices[left_merge_counter] < step_size/2);
                
                if (take_from_left) {
                    sorted_ints[dest_idx] = sorted_ints[left_idx];
                    merge_indices[left_merge_counter]++;
                    #ifdef DEBUG_SORT
                    if (threadIdx.x == print_thread_idx) {
                        printf("  Taking from left array\n");
                    }
                    #endif
                } else {
                    sorted_ints[dest_idx] = sorted_ints[right_idx]; 
                    merge_indices[right_merge_counter]++;
                    #ifdef DEBUG_SORT
                    if (threadIdx.x == print_thread_idx) {
                        printf("  Taking from right array\n");
                    }
                    #endif
                }
                
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("  New value at destination: %d\n", sorted_ints[dest_idx]);
                    printf("  Left merge index: %d, Right merge index: %d\n\n", 
                           merge_indices[left_merge_counter], merge_indices[right_merge_counter]);
                    printf("  Current array state: ");
                    for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                        printf("%d ", sorted_ints[j + parity_shift(!parity)]);
                    }
                    printf("\n");
                }
                #endif
            }
        }
        if (threadIdx.x == 0) {
            step_size *= 2;
            parity = !parity;
        }
        __syncthreads();
        #ifdef DEBUG_SORT
        if (threadIdx.x == 0) {
            printf("Random ints after step %d: ", step_size);
            for (int i = 0; i < PERMUTATION_LENGTH; i++) {
                printf("%d ", sorted_ints[i + parity_shift(parity)]);
            }
            printf("\n");
        }
        #endif
        __syncthreads();
    }
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Sorted random ints: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", sorted_ints[i + parity_shift(parity)]);
        }
        printf("\n");
    }
    #endif
    __syncthreads();
    int my_value = random_ints[threadIdx.x];
    int final_index = -1;
    for (int i = 0; i < PERMUTATION_LENGTH; i++) {
        if (sorted_ints[i + parity_shift(parity)] == my_value) {
            final_index = i;
            break;
        }
    }
    for (int i = 0; i < PERMUTATION_LENGTH; i++) {
        data[threadIdx.x * PERMUTATION_LENGTH + i] = (i == final_index) ? 1 : 0;
    }
    __syncthreads();

    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Final indices matrix:\n");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("  ");
            for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                printf("%d ", data[i * PERMUTATION_LENGTH + j]);
            }
            printf("\n");
        }
        printf("\n");
    }
    #endif

    // data[threadIdx.x] = sorted_ints[threadIdx.x + parity_shift(!parity)];
    __syncthreads();
}

__device__ void bogo_sort_permutation_gen(int* data, int size, int* random_ints) {
    extern __shared__ int sorted_ints[64];
    auto parity_shift = [](int p) { return p ? PERMUTATION_LENGTH : 0;};
    
    __syncthreads();
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Random ints: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", random_ints[i]);
        }
        printf("\n");
    }
    #endif
    __syncthreads();

    // Copy random ints to sorted_ints initial parity section
    sorted_ints[threadIdx.x] = random_ints[threadIdx.x];
    __syncthreads();

    extern __shared__ int step_size;
    extern __shared__ bool parity;
    extern __shared__ int merge_indices[PERMUTATION_LENGTH];
    if (threadIdx.x == 0) {
        step_size = 2;
        parity = false;
    }
    __syncthreads();

    while (step_size <= PERMUTATION_LENGTH) {
        if (threadIdx.x % step_size == 0) {
            int left_merge_counter = threadIdx.x;
            int right_merge_counter = threadIdx.x + 1;
            #ifdef DEBUG_SORT
            printf("Thread %d: left_merge_counter=%d, right_merge_counter=%d\n", 
                   threadIdx.x, left_merge_counter, right_merge_counter);
            #endif

            merge_indices[left_merge_counter] = 0;
            merge_indices[right_merge_counter] = 0;
            int print_thread_idx = 0;
            for (int i=0; i < step_size; i++) {
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("Thread %d, Step %d, Iteration %d:\n", threadIdx.x, step_size, i);
                }
                #endif
                
                int left_idx = threadIdx.x + merge_indices[left_merge_counter] + parity_shift(parity);
                int right_idx = threadIdx.x + merge_indices[right_merge_counter] + step_size/2 + parity_shift(parity);
                int dest_idx = threadIdx.x + i + parity_shift(!parity);
                
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("  Left index: %d (value: %d)\n", left_idx, sorted_ints[left_idx]);
                    printf("  Right index: %d (value: %d)\n", right_idx, sorted_ints[right_idx]);
                    printf("  Destination index: %d\n", dest_idx);
                }
                #endif
                
                bool take_from_left = merge_indices[right_merge_counter] == step_size/2 ||
                    (sorted_ints[left_idx] < sorted_ints[right_idx] && 
                     merge_indices[left_merge_counter] < step_size/2);
                
                if (take_from_left) {
                    sorted_ints[dest_idx] = sorted_ints[left_idx];
                    merge_indices[left_merge_counter]++;
                    #ifdef DEBUG_SORT
                    if (threadIdx.x == print_thread_idx) {
                        printf("  Taking from left array\n");
                    }
                    #endif
                } else {
                    sorted_ints[dest_idx] = sorted_ints[right_idx]; 
                    merge_indices[right_merge_counter]++;
                    #ifdef DEBUG_SORT
                    if (threadIdx.x == print_thread_idx) {
                        printf("  Taking from right array\n");
                    }
                    #endif
                }
                
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("  New value at destination: %d\n", sorted_ints[dest_idx]);
                    printf("  Left merge index: %d, Right merge index: %d\n\n", 
                           merge_indices[left_merge_counter], merge_indices[right_merge_counter]);
                    printf("  Current array state: ");
                    for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                        printf("%d ", sorted_ints[j + parity_shift(!parity)]);
                    }
                    printf("\n");
                }
                #endif
            }
        }
        if (threadIdx.x == 0) {
            step_size *= 2;
            parity = !parity;
        }
        __syncthreads();
        #ifdef DEBUG_SORT
        if (threadIdx.x == 0) {
            printf("Random ints after step %d: ", step_size);
            for (int i = 0; i < PERMUTATION_LENGTH; i++) {
                printf("%d ", sorted_ints[i + parity_shift(parity)]);
            }
            printf("\n");
        }
        #endif
        __syncthreads();
    }
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Sorted random ints: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", sorted_ints[i + parity_shift(parity)]);
        }
        printf("\n");
    }
    #endif
    __syncthreads();
    int my_value = random_ints[threadIdx.x];
    int my_index = -1;
    for (int i = 0; i < PERMUTATION_LENGTH; i++) {
        if (sorted_ints[i + parity_shift(parity)] == my_value) {
            my_index = i;
            break;
        }
    }
    data[threadIdx.x] = my_index;
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Final sorted array: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", data[i]);
        }
        printf("\n");
    }
    #endif
    __syncthreads();
}

dim3 KernelManagerBogoSortMatV1::calculateGrid(int N, int threadsPerBlock) {
    // return dim3((N + threadsPerBlock - 1) / threadsPerBlock);
    return dim3(N);
}

float KernelManagerBogoSortMatV1::launchKernel(int* data, int* output) {
    int size = 32;
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    int smCount = 0;
    for (int i = 0; i < deviceCount; i = i + 1) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        smCount += prop.multiProcessorCount;
    }

    #ifdef DEBUG_PRINT
    printf("Number of SMs: %d\n", smCount);
    #endif

    int numBlocks = smCount * 32;
    #ifdef DEBUG_PRINT
    printf("Number of blocks: %d\n", numBlocks);
    #endif

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int threadsPerBlock = 32;
    dim3 grid = calculateGrid(numBlocks, threadsPerBlock);
    dim3 block(threadsPerBlock);
    #ifdef DEBUG_PRINT
    printf("Grid dimensions: %d x %d x %d\n", grid.x, grid.y, grid.z);
    #endif

    // Record start time
    hipEventRecord(start);

    // Launch kernel
    // bogo_sort_matv1<<<grid, block>>>(data, size, output);
    bogo_sort_matv1<<<1, block>>>(data, size, output);

    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds;
}
