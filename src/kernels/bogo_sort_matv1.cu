#include "hip/hip_runtime.h"
#include "bogo_sort_matv1.cuh"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>
#include <mma.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>

// #define DEBUG_PERMUTE
#define DEBUG_PRINT 
// #define DEBUG_SORT
#define DEBUG_RANDOM
#define PERMUTE_MATRIX_WIDTH 16
#define PERMUTATION_LENGTH 32
#define PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024 1024
#define PERMUTATION_VECTORS_32x16_FLAT_LENGTH_512 512
#define LOWER_ROW 512
#define NEXT_BLOCK 16

#define OUTER_WIDTH 16
#define INNER_DIM 16
#define OUTER_HEIGHT 16

#define TOTAL_PERMUTATIONS 100
// #define TOTAL_PERMUTATIONS 10
#define CHECK_DONE_PERMUTATIONS 1000000

using namespace nvcuda;
using namespace std;

__global__ void bogo_sort_matv1(int* data, int size, int* output) {
    // extern __device__ int done;
    extern __shared__ int local_done;
    extern __shared__ bool is_sorted;

    if (threadIdx.x == 0) {
        local_done = 0;
        is_sorted = false;
        // if (blockIdx.x == 0) {
        //     done = 0;
        // }
    }
    __syncthreads();

    extern __shared__ __half permutation_matrix[PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024 * 2]; // two 32x32 arrays
    extern __shared__ __half permutation_vectors[PERMUTATION_VECTORS_32x16_FLAT_LENGTH_512];
    extern __shared__ int temp_permutation[PERMUTATION_LENGTH];

    #ifdef DEBUG_PERMUTE
    for (int i = 0; i < PERMUTE_MATRIX_WIDTH; i++) {
        data[i * PERMUTATION_LENGTH + threadIdx.x] = __float2half(threadIdx.x);
    }
    __syncthreads();
    #endif
    
    // Initialize random states and generate random ints
    extern __shared__ hiprandStatePhilox4_32_10_t random_states[PERMUTATION_LENGTH];
    extern __shared__ int random_ints[PERMUTATION_LENGTH];
    //hiprand_init(unsigned long long seed,
    // unsigned long long subsequence,
    // unsigned long long offset,
    // hiprandStatePhilox4_32_10_t *state)   
    hiprand_init(blockIdx.x, threadIdx.x, 0, &random_states[threadIdx.x]);
    __syncthreads();
    
    random_ints[threadIdx.x] = hiprand(&random_states[threadIdx.x]);
    bogo_sort_permutation_gen(temp_permutation, size, random_ints);

    for (int i = 0; i < PERMUTE_MATRIX_WIDTH; i++) {
        random_ints[threadIdx.x] = hiprand(&random_states[threadIdx.x]);
        __syncthreads();
        bogo_sort_permutation_gen(temp_permutation, size, random_ints);
        permutation_vectors[i * PERMUTATION_LENGTH + threadIdx.x] = __float2half(data[temp_permutation[threadIdx.x]]);
        __syncthreads();
    }

    random_ints[threadIdx.x] = hiprand(&random_states[threadIdx.x]);
    __syncthreads();
    bogo_sort_basis_gen(permutation_matrix, size, random_ints);
    __syncthreads();

    random_ints[threadIdx.x] = hiprand(&random_states[threadIdx.x]);
    __syncthreads();
    bogo_sort_basis_gen(permutation_matrix + PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024, size, random_ints);
    __syncthreads();

    #ifdef DEBUG_PRINT
    if (threadIdx.x == 0) {
        printf("Before Matmul Permutation vectors:\n");
        for (int i = 0; i < PERMUTE_MATRIX_WIDTH; i++) {
            printf("  Row %2d: ", i);
            for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                printf("%.1f ", __half2float(permutation_vectors[i * PERMUTATION_LENGTH + j]));
            }
            printf("\n");
        }
        printf("\n");
    }
    __syncthreads();
    #endif

    extern __shared__ long permutations_tried;
    extern __shared__ uint32_t switch_indexer;
    extern __shared__ uint32_t switch_multiplier;
    extern __shared__ uint32_t switch_incrementer;
    if (threadIdx.x == 0) {
        permutations_tried = 0;
        switch_indexer = hiprand(&random_states[threadIdx.x]);
        switch_incrementer = hiprand(&random_states[threadIdx.x]);
        switch_multiplier = switch_indexer;
    }
    __syncthreads();

    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_ne_frag;
    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_nw_frag;
    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_se_frag;
    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_sw_frag;

    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_ne_alt_frag;
    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_nw_alt_frag;
    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_se_alt_frag;
    wmma::fragment<wmma::matrix_a, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::row_major> mat_sw_alt_frag;

    wmma::fragment<wmma::matrix_b, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::col_major> vec_up_frag;
    wmma::fragment<wmma::matrix_b, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half, wmma::col_major> vec_down_frag;

    wmma::fragment<wmma::accumulator, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half> prod_up_frag;
    wmma::fragment<wmma::accumulator, OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT, half> prod_down_frag;

    if (threadIdx.x == 0) {
        printf("Reached tensor core initialization. Starting with %ld permutations tried.\n", permutations_tried);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        printf("Loading permutation vector fragments...\n");
        printf("Loading upper permutation vector fragment...\n");
        printf("  - Fragment type: matrix_b (input matrix B)\n");
        printf("  - Matrix layout: column-major\n");
        printf("  - Fragment dimensions: %dx%dx%d (MxNxK)\n", OUTER_WIDTH, INNER_DIM, OUTER_HEIGHT);
        printf("  - Data type: half precision (FP16)\n");
        printf("  - Source: permutation_vectors array\n");
        printf("  - Leading dimension: %d\n", PERMUTATION_LENGTH);
        printf("  - Loading %d elements starting at index 0\n", PERMUTATION_LENGTH);
        printf("\nPermutation vectors at load point:\n");
        printf("Upper vector (first 16 elements):\n");
        for (int i = 0; i < 16; i++) {
            printf("%.1f ", __half2float(permutation_vectors[i]));
            if ((i + 1) % 8 == 0) printf("\n");
        }
        printf("\nLower vector (first 16 elements):\n"); 
        for (int i = 0; i < 16; i++) {
            printf("%.1f ", __half2float(permutation_vectors[i + NEXT_BLOCK]));
            if ((i + 1) % 8 == 0) printf("\n");
        }

        printf("\nFragment contents after loading:\n");
        printf("vec_up_frag elements:\n");
        for (int i = 0; i < vec_up_frag.num_elements; i++) {
            printf("%.1f ", __half2float(vec_up_frag.x[i]));
            if ((i + 1) % 8 == 0) printf("\n");
        }
        printf("\nvec_down_frag elements:\n");
        for (int i = 0; i < vec_down_frag.num_elements; i++) {
            printf("%.1f ", __half2float(vec_down_frag.x[i]));
            if ((i + 1) % 8 == 0) printf("\n");
        }

    }
    wmma::load_matrix_sync(vec_up_frag, permutation_vectors, PERMUTATION_LENGTH);
    if (threadIdx.x == 0) {
        printf("Loading lower permutation vector fragment...\n");
    }
    wmma::load_matrix_sync(vec_down_frag, permutation_vectors + NEXT_BLOCK, PERMUTATION_LENGTH);

    if (threadIdx.x == 0) {
        printf("Loading primary permutation matrix fragments...\n"); 
    }
    wmma::load_matrix_sync(mat_nw_frag, permutation_matrix, PERMUTATION_LENGTH);
    wmma::load_matrix_sync(mat_ne_frag, permutation_matrix + NEXT_BLOCK, PERMUTATION_LENGTH);
    wmma::load_matrix_sync(mat_sw_frag, permutation_matrix + LOWER_ROW, PERMUTATION_LENGTH);
    wmma::load_matrix_sync(mat_se_frag, permutation_matrix + LOWER_ROW + NEXT_BLOCK, PERMUTATION_LENGTH);

    if (threadIdx.x == 0) {
        printf("Loading alternate permutation matrix fragments...\n");
    }
    wmma::load_matrix_sync(mat_nw_alt_frag, permutation_matrix + PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024, PERMUTATION_LENGTH);
    wmma::load_matrix_sync(mat_ne_alt_frag, permutation_matrix + NEXT_BLOCK + PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024, PERMUTATION_LENGTH);
    wmma::load_matrix_sync(mat_sw_alt_frag, permutation_matrix + LOWER_ROW + PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024, PERMUTATION_LENGTH);
    wmma::load_matrix_sync(mat_se_alt_frag, permutation_matrix + LOWER_ROW + NEXT_BLOCK + PERMUTATION_MATRIX_32x32_FLAT_LENGTH_1024, PERMUTATION_LENGTH);
    if (threadIdx.x == 0) {
        printf("Reached tensor core loading. Loaded all matrix fragments for permutation generation.\n");
    }
    __syncthreads();

    while (permutations_tried < TOTAL_PERMUTATIONS) {
        if (threadIdx.x == 0) {
            printf("Reached permutation %ld\n", permutations_tried);
        }
        // get 16th bit of switch_indexer
        bool random_bit = (switch_indexer >> 16) & 1;

        #ifdef DEBUG_RANDOM
        if (threadIdx.x == 0) {
            printf("switch_indexer:     ");
            for (int i = 31; i >= 0; i--) {
                printf("%d", (switch_indexer >> i) & 1);
                if (i % 8 == 0) printf(" ");
            }
            printf("\nswitch_incrementer: ");
            for (int i = 31; i >= 0; i--) {
                printf("%d", (switch_incrementer >> i) & 1);
                if (i % 8 == 0) printf(" ");
            }
            printf("\nswitch_multiplier:  ");
            for (int i = 31; i >= 0; i--) {
                printf("%d", (switch_multiplier >> i) & 1);
                if (i % 8 == 0) printf(" ");
            }
            printf("\nrandom_bit:         %s\n\n", random_bit ? "true" : "false");
        }
        #endif

        if (random_bit) {
            wmma::fill_fragment(prod_up_frag, 0.0f);
            wmma::mma_sync(prod_up_frag, mat_nw_frag, vec_up_frag, prod_up_frag);
            wmma::mma_sync(prod_up_frag, mat_ne_frag, vec_down_frag, prod_up_frag);
            wmma::store_matrix_sync(permutation_vectors, prod_up_frag, PERMUTATION_LENGTH, wmma::mem_col_major);

            wmma::fill_fragment(prod_down_frag, 0.0f);
            wmma::mma_sync(prod_down_frag, mat_sw_frag, vec_up_frag, prod_down_frag);
            wmma::mma_sync(prod_down_frag, mat_se_frag, vec_down_frag, prod_down_frag);
            wmma::store_matrix_sync(permutation_vectors + NEXT_BLOCK, prod_down_frag, PERMUTATION_LENGTH, wmma::mem_col_major);
        } else {
            wmma::fill_fragment(prod_up_frag, 0.0f);
            wmma::mma_sync(prod_up_frag, mat_nw_alt_frag, vec_up_frag, prod_up_frag);
            wmma::mma_sync(prod_up_frag, mat_ne_alt_frag, vec_down_frag, prod_up_frag);
            wmma::store_matrix_sync(permutation_vectors, prod_up_frag, PERMUTATION_LENGTH, wmma::mem_col_major);

            wmma::fill_fragment(prod_down_frag, 0.0f);
            wmma::mma_sync(prod_down_frag, mat_sw_alt_frag, vec_up_frag, prod_down_frag);
            wmma::mma_sync(prod_down_frag, mat_se_alt_frag, vec_down_frag, prod_down_frag);
            wmma::store_matrix_sync(permutation_vectors + NEXT_BLOCK, prod_down_frag, PERMUTATION_LENGTH, wmma::mem_col_major);
        }

        wmma::load_matrix_sync(vec_up_frag, permutation_vectors, PERMUTATION_LENGTH);
        wmma::load_matrix_sync(vec_down_frag, permutation_vectors + NEXT_BLOCK, PERMUTATION_LENGTH);

        if (threadIdx.x == 0) {
            // shift all bits in incrementer left by 1
            switch_incrementer = (switch_incrementer << 1) | (switch_incrementer >> 31);
            switch_indexer = switch_indexer * switch_multiplier + switch_incrementer;
            permutations_tried++;
            // printf("Reached switch update at permutation %d with indexer=0x%08x, incrementer=0x%08x, multiplier=0x%08x\n", 
            //        permutations_tried, switch_indexer, switch_incrementer, switch_multiplier);
            // printf("indexer: ");
            // for (int i = 31; i >= 0; i--) {
            //     printf("%d", (switch_indexer >> i) & 1);
            //     if (i % 8 == 0) printf(" ");
            // }
            // printf("\n");
        }

        __syncthreads();

        for (int i = 0; i < 16; i++) {
            verify_sort_matv1(permutation_vectors + i * 32, 32, &is_sorted);
            if (is_sorted) {
                output[threadIdx.x] = permutation_vectors[i * 32 + threadIdx.x];
                if (threadIdx.x == 0) {
                    printf("Block %d found sorted array after %ld permutations\n", blockIdx.x, permutations_tried);
                    // atomicCAS(&done, 0, 1);
                }
                return;
            }
        }

        if (permutations_tried % CHECK_DONE_PERMUTATIONS == 0) {
            if (threadIdx.x == 0) {
                // local_done = atomicAnd(&done, 1);
            }
            __syncthreads();
            // if (local_done) {
            //     if (blockIdx.x%100 ==0 && threadIdx.x == 0) {
            //         printf("Block %d: Permutations tried: %d\n", blockIdx.x, permutations_tried);
            //     }
            //     return;
            // }
        }
    }

    #ifdef DEBUG_PRINT
    if (threadIdx.x == 0) {
        printf("After Matmul Permutation vectors:\n");
        for (int i = 0; i < PERMUTE_MATRIX_WIDTH; i++) {
            printf("  Row %2d: ", i);
            for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                printf("%5.1f ", __half2float(permutation_vectors[i * PERMUTATION_LENGTH + j]));
            }
            printf("\n");
        }
        printf("\n");

        printf("Output data: ");
        for (int i = 0; i < size; i++) {
            printf("%d ", data[i]);
        }
        printf("\n");

        printf("Total permutations tried: %ld\n", permutations_tried);
    }
    #endif



    return;
}

__device__ void verify_sort_matv1(__half* input, int size, bool* is_sorted) {
    __syncthreads();
    if (threadIdx.x == 0) {
        *is_sorted = true;
    }
    __syncthreads();
    if (threadIdx.x < size - 1) {  // Don't check the last element since it has no right neighbor
        if (input[threadIdx.x] > input[threadIdx.x + 1]) {
            *is_sorted = false;
        }
    }
    __syncthreads();
}

__device__ void bogo_sort_basis_gen(__half* data, int size, int* random_ints) {
    extern __shared__ int sorted_ints[PERMUTATION_LENGTH * 2];
    auto parity_shift = [](int p) { return p ? PERMUTATION_LENGTH : 0;};
    
    __syncthreads();
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Random ints: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", random_ints[i]);
        }
        printf("\n");
    }
    #endif
    __syncthreads();

    // Copy random ints to sorted_ints initial parity section
    sorted_ints[threadIdx.x] = random_ints[threadIdx.x];
    __syncthreads();

    extern __shared__ int step_size;
    extern __shared__ bool parity;
    extern __shared__ int merge_indices[PERMUTATION_LENGTH];
    if (threadIdx.x == 0) {
        step_size = 2;
        parity = false;
    }
    __syncthreads();

    while (step_size <= PERMUTATION_LENGTH) {
        if (threadIdx.x % step_size == 0) {
            int left_merge_counter = threadIdx.x;
            int right_merge_counter = threadIdx.x + 1;
            #ifdef DEBUG_SORT
            printf("Thread %d: left_merge_counter=%d, right_merge_counter=%d\n", 
                   threadIdx.x, left_merge_counter, right_merge_counter);
            #endif

            merge_indices[left_merge_counter] = 0;
            merge_indices[right_merge_counter] = 0;
            int print_thread_idx = 0;
            for (int i=0; i < step_size; i++) {
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("Thread %d, Step %d, Iteration %d:\n", threadIdx.x, step_size, i);
                }
                #endif
                
                int left_idx = threadIdx.x + merge_indices[left_merge_counter] + parity_shift(parity);
                int right_idx = threadIdx.x + merge_indices[right_merge_counter] + step_size/2 + parity_shift(parity);
                int dest_idx = threadIdx.x + i + parity_shift(!parity);
                
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("  Left index: %d (value: %d)\n", left_idx, sorted_ints[left_idx]);
                    printf("  Right index: %d (value: %d)\n", right_idx, sorted_ints[right_idx]);
                    printf("  Destination index: %d\n", dest_idx);
                }
                #endif
                
                bool take_from_left = merge_indices[right_merge_counter] == step_size/2 ||
                    (sorted_ints[left_idx] < sorted_ints[right_idx] && 
                     merge_indices[left_merge_counter] < step_size/2);
                
                if (take_from_left) {
                    sorted_ints[dest_idx] = sorted_ints[left_idx];
                    merge_indices[left_merge_counter]++;
                    #ifdef DEBUG_SORT
                    if (threadIdx.x == print_thread_idx) {
                        printf("  Taking from left array\n");
                    }
                    #endif
                } else {
                    sorted_ints[dest_idx] = sorted_ints[right_idx]; 
                    merge_indices[right_merge_counter]++;
                    #ifdef DEBUG_SORT
                    if (threadIdx.x == print_thread_idx) {
                        printf("  Taking from right array\n");
                    }
                    #endif
                }
                
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("  New value at destination: %d\n", sorted_ints[dest_idx]);
                    printf("  Left merge index: %d, Right merge index: %d\n\n", 
                           merge_indices[left_merge_counter], merge_indices[right_merge_counter]);
                    printf("  Current array state: ");
                    for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                        printf("%d ", sorted_ints[j + parity_shift(!parity)]);
                    }
                    printf("\n");
                }
                #endif
            }
        }
        if (threadIdx.x == 0) {
            step_size *= 2;
            parity = !parity;
        }
        __syncthreads();
        #ifdef DEBUG_SORT
        if (threadIdx.x == 0) {
            printf("Random ints after step %d: ", step_size);
            for (int i = 0; i < PERMUTATION_LENGTH; i++) {
                printf("%d ", sorted_ints[i + parity_shift(parity)]);
            }
            printf("\n");
        }
        #endif
        __syncthreads();
    }
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Sorted random ints: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", sorted_ints[i + parity_shift(parity)]);
        }
        printf("\n");
    }
    #endif
    __syncthreads();
    int my_value = random_ints[threadIdx.x];
    int final_index = -1;
    for (int i = 0; i < PERMUTATION_LENGTH; i++) {
        if (sorted_ints[i + parity_shift(parity)] == my_value) {
            final_index = i;
            break;
        }
    }
    for (int i = 0; i < PERMUTATION_LENGTH; i++) {
        data[threadIdx.x * PERMUTATION_LENGTH + i] = __float2half(i == final_index ? 1.0f : 0.0f);
    }
    __syncthreads();

    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Final indices matrix:\n");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("  ");
            for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                printf("%d ", data[i * PERMUTATION_LENGTH + j]);
            }
            printf("\n");
        }
        printf("\n");
    }
    #endif

    // data[threadIdx.x] = sorted_ints[threadIdx.x + parity_shift(!parity)];
    __syncthreads();
}

__device__ void bogo_sort_permutation_gen(int* data, int size, int* random_ints) {
    extern __shared__ int sorted_ints[64];
    auto parity_shift = [](int p) { return p ? PERMUTATION_LENGTH : 0;};
    
    __syncthreads();
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Random ints: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", random_ints[i]);
        }
        printf("\n");
    }
    #endif
    __syncthreads();
    // Copy random ints to sorted_ints initial parity section
    sorted_ints[threadIdx.x] = random_ints[threadIdx.x];
    __syncthreads();
    extern __shared__ int step_size;
    extern __shared__ bool parity;
    extern __shared__ int merge_indices[PERMUTATION_LENGTH];
    if (threadIdx.x == 0) {
        step_size = 2;
        parity = false;
    }
    __syncthreads();
    while (step_size <= PERMUTATION_LENGTH) {
        if (threadIdx.x % step_size == 0) {
            int left_merge_counter = threadIdx.x;
            int right_merge_counter = threadIdx.x + 1;
            #ifdef DEBUG_SORT
            printf("Thread %d: left_merge_counter=%d, right_merge_counter=%d\n", 
                   threadIdx.x, left_merge_counter, right_merge_counter);
            #endif
            merge_indices[left_merge_counter] = 0;
            merge_indices[right_merge_counter] = 0;
            int print_thread_idx = 0;
            for (int i=0; i < step_size; i++) {
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("Thread %d, Step %d, Iteration %d:\n", threadIdx.x, step_size, i);
                }
                #endif
                
                int left_idx = threadIdx.x + merge_indices[left_merge_counter] + parity_shift(parity);
                int right_idx = threadIdx.x + merge_indices[right_merge_counter] + step_size/2 + parity_shift(parity);
                int dest_idx = threadIdx.x + i + parity_shift(!parity);
                
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("  Left index: %d (value: %d)\n", left_idx, sorted_ints[left_idx]);
                    printf("  Right index: %d (value: %d)\n", right_idx, sorted_ints[right_idx]);
                    printf("  Destination index: %d\n", dest_idx);
                }
                #endif
                
                bool take_from_left = merge_indices[right_merge_counter] == step_size/2 ||
                    (sorted_ints[left_idx] < sorted_ints[right_idx] && 
                     merge_indices[left_merge_counter] < step_size/2);
                
                if (take_from_left) {
                    sorted_ints[dest_idx] = sorted_ints[left_idx];
                    merge_indices[left_merge_counter]++;
                    #ifdef DEBUG_SORT
                    if (threadIdx.x == print_thread_idx) {
                        printf("  Taking from left array\n");
                    }
                    #endif
                } else {
                    sorted_ints[dest_idx] = sorted_ints[right_idx]; 
                    merge_indices[right_merge_counter]++;
                    #ifdef DEBUG_SORT
                    if (threadIdx.x == print_thread_idx) {
                        printf("  Taking from right array\n");
                    }
                    #endif
                }
                
                #ifdef DEBUG_SORT
                if (threadIdx.x == print_thread_idx) {
                    printf("  New value at destination: %d\n", sorted_ints[dest_idx]);
                    printf("  Left merge index: %d, Right merge index: %d\n\n", 
                           merge_indices[left_merge_counter], merge_indices[right_merge_counter]);
                    printf("  Current array state: ");
                    for (int j = 0; j < PERMUTATION_LENGTH; j++) {
                        printf("%d ", sorted_ints[j + parity_shift(!parity)]);
                    }
                    printf("\n");
                }
                #endif
            }
        }
        if (threadIdx.x == 0) {
            step_size *= 2;
            parity = !parity;
        }
        __syncthreads();
        #ifdef DEBUG_SORT
        if (threadIdx.x == 0) {
            printf("Random ints after step %d: ", step_size);
            for (int i = 0; i < PERMUTATION_LENGTH; i++) {
                printf("%d ", sorted_ints[i + parity_shift(parity)]);
            }
            printf("\n");
        }
        #endif
        __syncthreads();
    }
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Sorted random ints: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", sorted_ints[i + parity_shift(parity)]);
        }
        printf("\n");
    }
    #endif
    __syncthreads();
    int my_value = random_ints[threadIdx.x];
    int my_index = -1;
    for (int i = 0; i < PERMUTATION_LENGTH; i++) {
        if (sorted_ints[i + parity_shift(parity)] == my_value) {
            my_index = i;
            break;
        }
    }
    data[threadIdx.x] = my_index;
    #ifdef DEBUG_SORT
    if (threadIdx.x == 0) {
        printf("Final sorted array: ");
        for (int i = 0; i < PERMUTATION_LENGTH; i++) {
            printf("%d ", data[i]);
        }
        printf("\n");
    }
    #endif
    __syncthreads();
}


dim3 KernelManagerBogoSortMatV1::calculateGrid(int n, int threadsPerBlock) {
    // return dim3((INNER_DIM + threadsPerBlock - 1) / threadsPerBlock);
    return dim3(n);
}

float KernelManagerBogoSortMatV1::launchKernel(int* data, int* output) {
    int size = 32;
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    int smCount = 0;
    for (int i = 0; i < deviceCount; i = i + 1) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        smCount += prop.multiProcessorCount;
    }

    #ifdef DEBUG_PRINT
    printf("Number of SMs: %d\n", smCount);
    #endif

    int numBlocks = smCount * 64;
    #ifdef DEBUG_PRINT
    printf("Number of blocks: %d\n", numBlocks);
    #endif

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int threadsPerBlock = 32;
    dim3 grid = calculateGrid(numBlocks, threadsPerBlock);
    dim3 block(threadsPerBlock);
    #ifdef DEBUG_PRINT
    printf("Grid dimensions: %d x %d x %d\n", grid.x, grid.y, grid.z);
    #endif

    // Record start time
    hipEventRecord(start);

    // Launch kernel
    // bogo_sort_matv1<<<grid, block>>>(data, size, output);
    bogo_sort_matv1<<<1, block>>>(data, size, output);

    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds;
}
