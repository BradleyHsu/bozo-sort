#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../src/kernels/basic_kernel.cuh"
#include "../src/kernels/bogo_sort.cuh"

// Simple test framework
#define RUN_TEST(test_func) do { \
    printf("Running %s...\n", #test_func); \
    if (test_func()) { \
        printf("PASSED\n"); \
    } else { \
        printf("FAILED\n"); \
    } \
} while(0)

bool test_vector_add() {
    const int N = 1000;
    size_t size = N * sizeof(float);

    // Allocate and initialize host memory
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy inputs to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Run kernel
    KernelManagerVectorAdd::launchKernel(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    bool success = true;
    for (int i = 0; i < N; i++) {
        if (h_C[i] != 3.0f) {
            success = false;
            break;
        }
    }

    // Cleanup
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return success;
}

bool test_bogo_sort() {
    const int N = 32;
    size_t size = N * sizeof(int);

    // Allocate and initialize host memory
    int *h_input = new int[N];
    int *h_output = new int[N];
    
    // Initialize input array
    for (int i = 0; i < N; i++) {
        h_input[i] = i;
        h_output[i] = 0;
    }

    // Expected output array
    int expected[32] = {8,24,25,27,26,29,28,31,30,23,22,20,21,16,17,18,19,2,3,1,0,6,7,5,4,13,12,15,14,10,11,9};

    // Allocate device memory
    int *d_input, *d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    // Copy input to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
    hipMemcpy(d_output, h_output, size, hipMemcpyHostToDevice);

    // Run kernel
    KernelManagerBogoSort::launchKernel(d_input, N, d_output);

    // Copy result back to host
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    // Verify output matches expected array
    bool success = true;
    for (int i = 0; i < N; i++) {
        if (h_output[i] != expected[i]) {
            success = false;
            break;
        }
    }

    // Cleanup
    delete[] h_input;
    delete[] h_output;
    hipFree(d_input);
    hipFree(d_output);

    return success;

}
int main() {
    RUN_TEST(test_vector_add);
    RUN_TEST(test_bogo_sort);
    return 0;
}
